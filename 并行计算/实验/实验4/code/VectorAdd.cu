#include <stdio.h>
#include<hip/hip_runtime.h>
 
//__global__声明的函数，告诉编译器这段代码交由CPU调用，由GPU执行
__global__ void add(const int *dev_a, const int *dev_b, int *dev_c) {
    int i = threadIdx.x;
    dev_c[i] = dev_a[i] + dev_b[i];
}
 
int main(void) {
    //申请主机内存，并进行初始化
    int host_a[512], host_b[512], host_c[512];
    for(int i = 0; i < 512; i++) {
        host_a[i] = i;
        host_b[i] = i << 1;
    }
 
    //定义cudaError，默认为cudaSuccess(0)
    hipError_t err = hipSuccess;
 
    //申请GPU存储空间
    int *dev_a, *dev_b, *dev_c;
    err = hipMalloc((void **)&dev_a, sizeof(int) * 512);
    err = hipMalloc((void **)&dev_b, sizeof(int) * 512);
    err = hipMalloc((void **)&dev_c, sizeof(int) * 512);
    if(err != hipSuccess) {
        printf("the hipMalloc on GPU is failed");
        return 1;
    }
    printf("SUCCESS");
    //将要计算的数据使用cudaMemcpy传送到GPU
    hipMemcpy(dev_a, host_a, sizeof(host_a), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, sizeof(host_b), hipMemcpyHostToDevice);
 
    //调用核函数在GPU上执行。数据较少，只使用一个Block，含有512个线程
    add<<<1,512>>>(dev_a, dev_b, dev_c);
    hipMemcpy(&host_c, dev_c, sizeof(host_c), hipMemcpyDeviceToHost);
    for(int i = 0; i < 512; i++) {
        printf("host_a[%d] + host_b[%d] = %d + %d = %d\n", i, i, host_a[i], host_b[i], host_c[i]);
    }
    
    hipFree(dev_a);//释放GPU内存
    hipFree(dev_b);//释放GPU内存
    hipFree(dev_c);//释放GPU内存
    return 0 ;
}
