#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define N 1000

bool InitCUDA();
void matgen(float* a, int lda, int n);
clock_t matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
void compare_mat(const float* a, int lda, const float* b, int ldb, int n);
clock_t matmultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
__global__ static void matMultCUDA(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n);

int main() {
	float *a, *b, *c, *d;
	if (!InitCUDA()) return 0;
	a = (float*)malloc(sizeof(float) * N * N);
	b = (float*)malloc(sizeof(float) * N * N);
	c = (float*)malloc(sizeof(float) * N * N);
	d = (float*)malloc(sizeof(float) * N * N);
	srand(0);
	matgen(a, N, N);
	matgen(b, N, N);
	clock_t time1 = matmultCUDA(a, N, b, N, c, N, N);
	clock_t time2 = matmult(a, N, b, N, d, N, N);
	compare_mat(c, N, d, N, N);
	double sec1 = (double)time1 / CLOCKS_PER_SEC;
	double sec2 = (double)time2 / CLOCKS_PER_SEC;
	printf("Time used: %.6f seconds(%.2lf GFLOPS) in CUDA,Time used: %.6f seconds(%.2lf GFLOPS) in CPU\n", sec1, 2.0 * N * N * N / (sec1 * 1E9), sec2, 2.0 * N * N * N / (sec2 * 1E9));
	return 0;
}

// Initialize CUDA
bool InitCUDA() {
	int count;
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}

// Generate matrix data
void matgen(float* a, int lda, int n) {
	int i, j;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			a[i * lda + j] = (float)rand() / RAND_MAX;
		}
	}
}

// CPU version of Matrix Multiplication
clock_t matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n) {
	clock_t start, end;
	int i, j, k;
	start = clock();
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			double t = 0;
			for (k = 0; k < n; k++) {
				t += a[i * lda + k] * b[k * ldb + j];
			}
			c[i * ldc + j] = t;
		}
	}
	end = clock();
	return end - start;
}

// Compare Matrix A and Matrix B, and get the error value
void compare_mat(const float* a, int lda, const float* b, int ldb, int n) {
	float max_err = 0;
	float average_err = 0;
	int i, j;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			if (b[i * ldb + j] != 0) {
				float err = fabs((a[i * lda + j] - b[i * ldb + j]) / b[i * ldb + j]);
				if (max_err < err) max_err = err;
				average_err += err;
			}
		}
	}
	printf("Max error: %g Average error: %g\n", max_err, average_err / (n * n));
}

// Prepare for the CUDA version of Matrix Multiplication
clock_t matmultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n) {
	float *ac, *bc, *cc;
	clock_t start, end;
	
	start = clock();

	// Pitch memory for better performance
	size_t pitch_a, pitch_b, pitch_c;
	hipMallocPitch((void**)&ac, &pitch_a, sizeof(float) * n, n);
	hipMallocPitch((void**)&bc, &pitch_b, sizeof(float) * n, n);
	hipMallocPitch((void**)&cc, &pitch_c, sizeof(float) * n, n);

	// Copy the inputs, Matrix A and Matrix B, from host to deivce
	hipMemcpy2D(ac, pitch_a, a, sizeof(float) * lda, sizeof(float) * n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, pitch_b, b, sizeof(float) * ldb, sizeof(float) * n, n, hipMemcpyHostToDevice);

	//int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
	matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>(ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float), cc, pitch_c / sizeof(float), n);

	// Copy the result back from device to host
	hipMemcpy2D(c, sizeof(float) * ldc, cc, pitch_c, sizeof(float) * n, n, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(ac);
	hipFree(bc);
	hipFree(cc);
	end = clock();
	return end - start;
}

// CUDA version of Matrix Multiplication
__global__ static void matMultCUDA(const float* a, size_t lda,const float* b, size_t ldb, float* c, size_t ldc, int n) {
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;
	// load data from global memory to shared memory
	for (i = tid; i < n; i += blockDim.x) {
		data[i] = a[row * lda + i];
	}
	__syncthreads(); // barrier! make sure all threads are ready to avoid potential data conflict.

	// all threads in a block process one line of output
	for (j = tid; j < n; j += blockDim.x) {
		double y = 0;
		for (i = 0; i < n; i++) {
			y += data[i] * b[i * ldb + j];
		}
		c[row * ldc + j] = y;
	}
}
